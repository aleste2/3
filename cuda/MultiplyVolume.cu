#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange.go for more details.
extern "C" __global__ void
MultiplyVolume(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float volume,
            int N
            ) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        Bx[i] = volume*Bx[i];
        By[i] = volume*By[i];
        Bz[i] = volume*Bz[i];
    }
}