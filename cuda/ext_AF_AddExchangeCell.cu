#include "hip/hip_runtime.h"
#include "amul.h"
#include "float3.h"
#include <stdint.h>

// normalize vector {vx, vy, vz} to unit length, unless length or vol are zero.
extern "C" __global__ void
addExchangeAFCell(float* __restrict__ dst1x, float* __restrict__ dst1y, float* __restrict__ dst1z,
	float* __restrict__ dst2x, float* __restrict__ dst2y, float* __restrict__ dst2z,
	float* __restrict__ m1x, float* __restrict__ m1y, float* __restrict__ m1z,
	float* __restrict__ m2x, float* __restrict__ m2y, float* __restrict__ m2z,
	float* __restrict__  Ms1_, float  Ms1_mul,
	float* __restrict__  Ms2_, float  Ms2_mul,
	float* __restrict__  Bex12_, float  Bex12_mul,
    	float* __restrict__  Bex21_, float  Bex21_mul,
	int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float invMs1 = inv_Msat(Ms1_, Ms1_mul, i);
        float invMs2 = inv_Msat(Ms2_, Ms2_mul, i);
        float bex12 = amul(Bex12_, Bex12_mul, i);
        float bex21 = amul(Bex21_, Bex21_mul, i);

        dst1x[i] += invMs1*bex12*m2x[i];
        dst1y[i] += invMs1*bex12*m2y[i];
        dst1z[i] += invMs1*bex12*m2z[i];
        dst2x[i] += invMs2*bex21*m1x[i];
        dst2y[i] += invMs2*bex21*m1y[i];
        dst2z[i] += invMs2*bex21*m1z[i];
    }
}
