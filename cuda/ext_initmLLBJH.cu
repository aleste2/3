#include "hip/hip_runtime.h"
#include "amul.h"
#include "float3.h"
#include <stdint.h>

// Landau-Lifshitz torque.
extern "C" 

__global__ void
initmLLBJH(float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
        float* __restrict__  tempJH,
        float* __restrict__  TCurie_, float TCurie_mul,
        int N,int Langevin) {
 
    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if (i < N) {
        float3 m = {mx[i], my[i], mz[i]};
        float TCurie = amul(TCurie_, TCurie_mul, i);
        float temp = tempJH[i]; 
        if (temp==0) temp=0.0001; // to avoid zero division...
        float m2=dot(m,m);
 
        if ((m2!=0)&&(TCurie!=0))
        {
	 if (temp<=TCurie)  // T<Tc
         {
         	float me=pow(1.0f-pow(temp/TCurie,3.49f),0.54f);
		mx[i]=mx[i]*(me/pow(m2,0.5f));
		my[i]=my[i]*(me/pow(m2,0.5f));
		mz[i]=mz[i]*(me/pow(m2,0.5f));

         }
         else        //T>Tc
         {
         	float me=0.0001;
		mx[i]=mx[i]*(me/pow(m2,0.5f));
		my[i]=my[i]*(me/pow(m2,0.5f));
		mz[i]=mz[i]*(me/pow(m2,0.5f));

         };
	}
    }
}
